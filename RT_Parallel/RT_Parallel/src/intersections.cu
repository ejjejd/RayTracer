#include "hip/hip_runtime.h"
#define RT_DLL
#include <fstream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#include "include/intersections.h"

namespace cuda
{
	__device__ bool Intersection(Sphere* sphere, const Ray& ray, float* t)
	{
		float to = Dot(sphere->Center - ray.Origin, ray.Direction);	//Find the length to the sphere
		Vector3 point = PointAt(ray, to);							//Find intersection point
		float y = Length(sphere->Center - point);					//Find the length from intersect point to sphere center

		if (y > sphere->Radius)
			return false;

		//Find the nearest intersection
		float x = sqrt(sphere->Radius * sphere->Radius - y * y);
		*t = *t - x;

		if (*t < 0.0f)
		{
			*t = *t + x;
			if (*t < 0.0f)
				return false;
		}

		return true;
	}

	__device__ Vector3 GenerateDirection(const uint16_t x, const uint16_t y, const uint16_t sizeX, const uint16_t sizeY)
	{
		Vector3 ndc;
		ndc.x = (x + 0.5f) / static_cast<float>(sizeX);
		ndc.y = (y + 0.5f) / static_cast<float>(sizeY);

		Vector3 pixel;
		pixel.x = (2.0f * ndc.x - 1.0f) * (static_cast<float>(sizeX) / sizeY) * tan(Radians(45.0f / 2.0f));
		pixel.y = (1.0f - 2.0f * ndc.y) * tan(Radians(45.0f / 2.0f));
		pixel.z = -1.0f;

		return Normalize(pixel);
	}

	__global__ void FillPixel(Sphere* sphere, Vector3* cameraPosition, const uint16_t sizeX, const uint16_t sizeY, Vector3* framebuffer)
	{
		uint16_t x = (blockIdx.x * blockDim.x) + threadIdx.x;
		uint16_t y = (blockIdx.y * blockDim.y) + threadIdx.y;

		Ray ray;
		ray.Origin	  = *cameraPosition;
		ray.Direction = GenerateDirection(x, y, sizeX, sizeY);

		float t = INFINITY;

		if (Intersection(sphere, ray, &t))
			framebuffer[x + y * sizeX] = Vector3(1.0f, 0.2f, 0.2f);
		else
			framebuffer[x + y * sizeX] = Vector3(0.5f, 0.5f, 0.5f);
	}

	void Render(Sphere* sphere, Vector3 cameraPosition, const uint16_t sizeX, const uint16_t sizeY, Vector3* frameBuffer)
	{
		dim3 threadsPerBlock(16, 16);
		dim3 numBlocks(sizeX / threadsPerBlock.x, sizeY / threadsPerBlock.y);

		Vector3*	d_Framebuffer;
		Vector3*	d_CameraPos;
		Sphere*		d_Sphere;

		hipMalloc(reinterpret_cast<void**>(&d_Sphere), sizeof(Sphere));
		hipMalloc(reinterpret_cast<void**>(&d_Framebuffer), sizeof(Vector3) * sizeX * sizeY);
		hipMalloc(reinterpret_cast<void**>(&d_CameraPos), sizeof(Vector3));

		hipMemcpy(d_Sphere, sphere, sizeof(Sphere), hipMemcpyHostToDevice);
		hipMemcpy(d_CameraPos, &cameraPosition, sizeof(Vector3), hipMemcpyHostToDevice);

		FillPixel<<<numBlocks, threadsPerBlock>>>(d_Sphere, d_CameraPos, sizeX, sizeY, d_Framebuffer);

		hipMemcpy(frameBuffer, d_Framebuffer, sizeof(Vector3) * sizeX * sizeY, hipMemcpyDeviceToHost);

		hipFree(d_Framebuffer);
		hipFree(d_CameraPos);
		hipFree(d_Sphere);

		size_t freeBytes;
		size_t totalBytes;
		hipMemGetInfo(&freeBytes, &totalBytes);
		fprintf(stderr, "Used: %f, Free: %f\n", (float)freeBytes, (float)totalBytes);
	}
}

//int main()
//{
//	uint16_t sizeX = 1280;
//	uint16_t sizeY = 720;
//	
//	cuda::Sphere sphere = { cuda::Vector3(0.0f, 0.0f, -10.0f), 2.5f };
//	cuda::Vector3* Framebuffer = new cuda::Vector3[sizeX * sizeY];
//
//	cuda::Vector3 cp(10.0f, 10.0f, 10.f);
//
//	cuda::Render(&sphere, cp, sizeX, sizeY, Framebuffer);
//
//	//Write to file
//	std::ofstream outFile;
//	outFile.open("out.ppm");
//	if (!outFile.is_open())
//		return -1;
//
//	outFile << "P6\n" << sizeX << " " << sizeY << "\n255\n";
//	for (int i = 0; i < sizeX * sizeY; ++i)
//	{
//		outFile << (char)(255.0f * std::fmaxf(0.0f, std::fminf(1.0f, Framebuffer[i].x)));
//		outFile << (char)(255.0f * std::fmaxf(0.0f, std::fminf(1.0f, Framebuffer[i].y)));
//		outFile << (char)(255.0f * std::fmaxf(0.0f, std::fminf(1.0f, Framebuffer[i].z)));
//	}
//
//	return 0;
//}